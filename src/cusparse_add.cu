#include "hip/hip_runtime.h"
//
// Created by Egor.Orachev on 27.01.2021.
//

#include <benchmark_base.hpp>
#include <matrix_loader.hpp>
#include <args_processor.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <hipsparse.h>

#define CUSPARSE_CHECH(function) { auto statusCall = function; assert(statusCall == HIPSPARSE_STATUS_SUCCESS); }

#define BENCH_DEBUG

namespace benchmark {

    struct CsrMatrix {
        CsrMatrix() = default;

        CsrMatrix(const CsrMatrix &other) = default;

        CsrMatrix(CsrMatrix &&other) = default;

        CsrMatrix &operator=(const CsrMatrix &other) = default;

        CsrMatrix &operator=(CsrMatrix &&other) = default;

        void release() {
            rows.clear();
            cols.clear();
            nvals = 0;
            n = 0;
        }

        hipsparseMatDescr_t desc{};
        thrust::device_vector<int> rows;
        thrust::device_vector<int> cols;
        int nvals = 0;
        int n = 0;
    };

    class Add : public BenchmarkBase {
    public:

        Add(int argc, const char **argv) {
            argsProcessor.parse(argc, argv);
            assert(argsProcessor.isParsed());

            benchmarkName = "cuSPARSE-Add";
            experimentsCount = argsProcessor.getExperimentsCount();
        }

    protected:

        void setupBenchmark() override {
            CUSPARSE_CHECH(hipsparseCreate(&handle));
        }

        void tearDownBenchmark() override {
            hipsparseDestroy(handle);
            handle = nullptr;
        }

        void setupExperiment(size_t experimentIdx, size_t &iterationsCount) override {
            auto &entry = argsProcessor.getEntries()[experimentIdx];

            iterationsCount = entry.iterations;

            const auto &file = entry.name;
            const auto &type = entry.isUndirected;

            MatrixLoader loader(file, type);
            loader.loadData();
            input = std::move(loader.getMatrix());

#ifdef BENCH_DEBUG
            std::cout << ">   Load matrix: \"" << file << "\" isUndirected: " << type << std::endl
                      << "                 size: " << input.nrows << " x " << input.ncols << " nvals: " << input.nvals
                      << std::endl;
#endif // BENCH_DEBUG

            size_t n = input.nrows;
            assert(input.nrows == input.ncols);

            thrust::host_vector<int> rowsPtr(n + 1, 0);
            thrust::host_vector<int> colsInd(input.nvals);

            for (auto i = 0; i < input.nvals; i++) {
                rowsPtr[input.rows[i]] += 1;
                colsInd[i] = input.cols[i];
            }

            int sum = 0;
            for (auto &r: rowsPtr) {
                int prev = sum;
                sum += r;
                r = prev;
            }

            CUSPARSE_CHECH(hipsparseCreateMatDescr(&A.desc));
            CUSPARSE_CHECH(hipsparseSetMatType(A.desc, HIPSPARSE_MATRIX_TYPE_GENERAL));
            CUSPARSE_CHECH(hipsparseSetMatIndexBase(A.desc, HIPSPARSE_INDEX_BASE_ZERO));

            A.nvals = input.nvals;
            A.n = n;
            A.rows.resize(rowsPtr.size());
            A.cols.resize(colsInd.size());
            values.resize(A.nvals, 1.0f);

            thrust::copy(rowsPtr.begin(), rowsPtr.end(), A.rows.begin());
            thrust::copy(colsInd.begin(), colsInd.end(), A.cols.begin());

            // A2 = A * A

            CUSPARSE_CHECH(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

            CUSPARSE_CHECH(hipsparseCreateMatDescr(&A2.desc));
            CUSPARSE_CHECH(hipsparseSetMatType(A2.desc, HIPSPARSE_MATRIX_TYPE_GENERAL));
            CUSPARSE_CHECH(hipsparseSetMatIndexBase(A2.desc, HIPSPARSE_INDEX_BASE_ZERO));

            A2.rows.resize(A.n + 1);

            int nnzA2 = 0;
            int *nnzTotalDevHostPtr = &nnzA2;

            CUSPARSE_CHECH(hipsparseXcsrgemmNnz(
                    handle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    A.n, A.n, A.n,
                    A.desc, A.nvals, A.rows.data().get(), A.cols.data().get(),
                    A.desc, A.nvals, A.rows.data().get(), A.cols.data().get(),
                    A2.desc, A2.rows.data().get(), nnzTotalDevHostPtr
            ));

            assert(nnzTotalDevHostPtr != nullptr);
            nnzA2 = *nnzTotalDevHostPtr;

            A2.n = A.n;
            A2.nvals = nnzA2;
            A2.cols.resize(nnzA2);

            valuesA2.resize(nnzA2, 1.0f);

            CUSPARSE_CHECH(hipsparseScsrgemm(
                    handle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    A.n, A.n, A.n,
                    A.desc, A.nvals, values.data().get(), A.rows.data().get(), A.cols.data().get(),
                    A.desc, A.nvals, values.data().get(), A.rows.data().get(), A.cols.data().get(),
                    A2.desc, valuesA2.data().get(), A2.rows.data().get(), A2.cols.data().get()
            ));
        }

        void tearDownExperiment(size_t experimentIdx) override {
            input = Matrix{};

            CUSPARSE_CHECH(hipsparseDestroyMatDescr(A.desc));
            A.release();

            CUSPARSE_CHECH(hipsparseDestroyMatDescr(A2.desc));
            A2.release();
        }

        void setupIteration(size_t experimentIdx, size_t iterationIdx) override {
            CUSPARSE_CHECH(hipsparseCreateMatDescr(&R.desc));
            CUSPARSE_CHECH(hipsparseSetMatType(R.desc, HIPSPARSE_MATRIX_TYPE_GENERAL));
            CUSPARSE_CHECH(hipsparseSetMatIndexBase(R.desc, HIPSPARSE_INDEX_BASE_ZERO));

            R.rows.resize(A.n + 1);
        }

        void execIteration(size_t experimentIdx, size_t iterationIdx) override {
            CUSPARSE_CHECH(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

            int nnzC = 0;
            int *nnzTotalDevHostPtr = &nnzC;

            CUSPARSE_CHECH(hipsparseXcsrgeamNnz(
                    handle,
                    A.n, A.n,
                    A.desc, A.nvals, A.rows.data().get(), A.cols.data().get(),
                    A2.desc, A2.nvals, A2.rows.data().get(), A2.cols.data().get(),
                    R.desc, R.rows.data().get(), nnzTotalDevHostPtr
            ));

            assert(nnzTotalDevHostPtr != nullptr);
            nnzC = *nnzTotalDevHostPtr;

            R.n = A.n;
            R.nvals = nnzC;
            R.cols.resize(nnzC);

            float alpha = 1.0f;
            float beta = 1.0f;

            thrust::device_vector<float> tmp(nnzC);

            CUSPARSE_CHECH(hipsparseScsrgeam(
                    handle,
                    A.n, A.n,
                    &alpha,
                    A.desc, A.nvals, values.data().get(), A.rows.data().get(), A.cols.data().get(),
                    &beta,
                    A2.desc, A2.nvals, valuesA2.data().get(), A2.rows.data().get(), A2.cols.data().get(),
                    R.desc, tmp.data().get(), R.rows.data().get(), R.cols.data().get()
            ));
        }

        void tearDownIteration(size_t experimentIdx, size_t iterationIdx) override {
#ifdef BENCH_DEBUG
            log << "   Result matrix: size " << R.n << " x " << R.n
                << " nvals " << R.nvals << std::endl;
#endif

            CUSPARSE_CHECH(hipsparseDestroyMatDescr(R.desc));
            R.release();
        }

    protected:

        hipsparseHandle_t handle;

        CsrMatrix A;
        CsrMatrix A2;
        CsrMatrix R;
        thrust::device_vector<float> values;
        thrust::device_vector<float> valuesA2;

        ArgsProcessor argsProcessor;
        Matrix input;

    };

}

int main(int argc, const char **argv) {
    benchmark::Add add(argc, argv);
    add.runBenchmark();
    return 0;
}