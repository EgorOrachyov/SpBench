#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////
// MIT License                                                                    //
//                                                                                //
// Copyright (c) 2021 Egor Orachyov                                               //
//                                                                                //
// Permission is hereby granted, free of charge, to any person obtaining a copy   //
// of this software and associated documentation files (the "Software"), to deal  //
// in the Software without restriction, including without limitation the rights   //
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      //
// copies of the Software, and to permit persons to whom the Software is          //
// furnished to do so, subject to the following conditions:                       //
//                                                                                //
// The above copyright notice and this permission notice shall be included in all //
// copies or substantial portions of the Software.                                //
//                                                                                //
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     //
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       //
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    //
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         //
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  //
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  //
// SOFTWARE.                                                                      //
////////////////////////////////////////////////////////////////////////////////////

#include <benchmark_base.hpp>
#include <matrix_loader.hpp>
#include <args_processor.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <hipsparse.h>

#define CUSPARSE_CHECH(function) { auto statusCall = function; assert(statusCall == HIPSPARSE_STATUS_SUCCESS); }

#define BENCH_DEBUG

namespace benchmark {

    struct CsrMatrix {
        CsrMatrix() = default;
        CsrMatrix(const CsrMatrix &other) = default;
        CsrMatrix(CsrMatrix &&other) = default;

        CsrMatrix &operator=(const CsrMatrix &other) = default;
        CsrMatrix &operator=(CsrMatrix &&other) = default;

        void release() {
            rows.clear();
            cols.clear();
            nvals = 0;
            n = 0;
        }

        hipsparseMatDescr_t desc{};
        thrust::device_vector<int> rows;
        thrust::device_vector<int> cols;
        int nvals = 0;
        int n = 0;
    };

    class Add : public BenchmarkBase {
    public:

        Add(int argc, const char **argv) {
            argsProcessor.parse(argc, argv);
            assert(argsProcessor.isParsed());

            benchmarkName = "cuSPARSE-Add";
            experimentsCount = argsProcessor.getExperimentsCount();
        }

    protected:

        void setupBenchmark() override {
            CUSPARSE_CHECH(hipsparseCreate(&handle));
        }

        void tearDownBenchmark() override {
            hipsparseDestroy(handle);
            handle = nullptr;
        }

        void setupExperiment(size_t experimentIdx, size_t &iterationsCount, std::string& name) override {
            auto &entry = argsProcessor.getEntries()[experimentIdx];

            iterationsCount = entry.iterations;
            name = entry.name;

            const auto &file = entry.name;
            const auto &type = entry.isUndirected;

            MatrixLoader loader(file, type);
            loader.loadData();
            input = std::move(loader.getMatrix());

#ifdef BENCH_DEBUG
            log       << ">   Load A: \"" << file << "\" isUndirected: " << type << std::endl
                      << "                 size: " << input.nrows << " x " << input.ncols << " nvals: " << input.nvals
                      << std::endl;
#endif // BENCH_DEBUG

            {
                size_t n = input.nrows;
                assert(input.nrows == input.ncols);

                thrust::host_vector<int> rowsPtr(n + 1, 0);
                thrust::host_vector<int> colsInd(input.nvals);

                for (auto i = 0; i < input.nvals; i++) {
                    rowsPtr[input.rows[i]] += 1;
                    colsInd[i] = input.cols[i];
                }

                int sum = 0;
                for (auto &r: rowsPtr) {
                    int prev = sum;
                    sum += r;
                    r = prev;
                }

                CUSPARSE_CHECH(hipsparseCreateMatDescr(&A.desc));
                CUSPARSE_CHECH(hipsparseSetMatType(A.desc, HIPSPARSE_MATRIX_TYPE_GENERAL));
                CUSPARSE_CHECH(hipsparseSetMatIndexBase(A.desc, HIPSPARSE_INDEX_BASE_ZERO));

                A.nvals = input.nvals;
                A.n = n;
                A.rows.resize(rowsPtr.size());
                A.cols.resize(colsInd.size());
                values.resize(A.nvals, 1.0f);

                thrust::copy(rowsPtr.begin(), rowsPtr.end(), A.rows.begin());
                thrust::copy(colsInd.begin(), colsInd.end(), A.cols.begin());
            }


            MatrixLoader2 loader2(file);
            loader2.loadData();
            input = std::move(loader2.getMatrix());

#ifdef BENCH_DEBUG
            log       << ">   Load A2: \"" << file << "\" isUndirected: " << type << std::endl
                      << "                 size: " << input.nrows << " x " << input.ncols << " nvals: " << input.nvals
                      << std::endl;
#endif // BENCH_DEBUG

            {
                size_t n = input.nrows;
                assert(input.nrows == input.ncols);

                thrust::host_vector<int> rowsPtr(n + 1, 0);
                thrust::host_vector<int> colsInd(input.nvals);

                for (auto i = 0; i < input.nvals; i++) {
                    rowsPtr[input.rows[i]] += 1;
                    colsInd[i] = input.cols[i];
                }

                int sum = 0;
                for (auto &r: rowsPtr) {
                    int prev = sum;
                    sum += r;
                    r = prev;
                }

                CUSPARSE_CHECH(hipsparseCreateMatDescr(&A2.desc));
                CUSPARSE_CHECH(hipsparseSetMatType(A2.desc, HIPSPARSE_MATRIX_TYPE_GENERAL));
                CUSPARSE_CHECH(hipsparseSetMatIndexBase(A2.desc, HIPSPARSE_INDEX_BASE_ZERO));

                A2.nvals = input.nvals;
                A2.n = n;
                A2.rows.resize(rowsPtr.size());
                A2.cols.resize(colsInd.size());
                valuesA2.resize(A2.nvals, 1.0f);

                thrust::copy(rowsPtr.begin(), rowsPtr.end(), A2.rows.begin());
                thrust::copy(colsInd.begin(), colsInd.end(), A2.cols.begin());
            }
        }

        void tearDownExperiment(size_t experimentIdx) override {
            input = Matrix{};

            CUSPARSE_CHECH(hipsparseDestroyMatDescr(A.desc));
            A.release();

            CUSPARSE_CHECH(hipsparseDestroyMatDescr(A2.desc));
            A2.release();
        }

        void setupIteration(size_t experimentIdx, size_t iterationIdx) override {
            CUSPARSE_CHECH(hipsparseCreateMatDescr(&R.desc));
            CUSPARSE_CHECH(hipsparseSetMatType(R.desc, HIPSPARSE_MATRIX_TYPE_GENERAL));
            CUSPARSE_CHECH(hipsparseSetMatIndexBase(R.desc, HIPSPARSE_INDEX_BASE_ZERO));

            R.rows.resize(A.n + 1);
        }

        void execIteration(size_t experimentIdx, size_t iterationIdx) override {
            CUSPARSE_CHECH(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

            int nnzC = 0;
            int *nnzTotalDevHostPtr = &nnzC;

            CUSPARSE_CHECH(hipsparseXcsrgeamNnz(
                    handle,
                    A.n, A.n,
                    A.desc, A.nvals, A.rows.data().get(), A.cols.data().get(),
                    A2.desc, A2.nvals, A2.rows.data().get(), A2.cols.data().get(),
                    R.desc, R.rows.data().get(), nnzTotalDevHostPtr
            ));

            assert(nnzTotalDevHostPtr != nullptr);
            nnzC = *nnzTotalDevHostPtr;

            R.n = A.n;
            R.nvals = nnzC;
            R.cols.resize(nnzC);

            float alpha = 1.0f;
            float beta = 1.0f;

            thrust::device_vector<float> tmp(nnzC);

            CUSPARSE_CHECH(hipsparseScsrgeam(
                    handle,
                    A.n, A.n,
                    &alpha,
                    A.desc, A.nvals, values.data().get(), A.rows.data().get(), A.cols.data().get(),
                    &beta,
                    A2.desc, A2.nvals, valuesA2.data().get(), A2.rows.data().get(), A2.cols.data().get(),
                    R.desc, tmp.data().get(), R.rows.data().get(), R.cols.data().get()
            ));
        }

        void tearDownIteration(size_t experimentIdx, size_t iterationIdx) override {
#ifdef BENCH_DEBUG
            log << "   Result matrix: size " << R.n << " x " << R.n
                << " nvals " << R.nvals << std::endl;
#endif

            CUSPARSE_CHECH(hipsparseDestroyMatDescr(R.desc));
            R.release();
        }

    protected:

        hipsparseHandle_t handle;

        CsrMatrix A;
        CsrMatrix A2;
        CsrMatrix R;
        thrust::device_vector<float> values;
        thrust::device_vector<float> valuesA2;

        ArgsProcessor argsProcessor;
        Matrix input;

    };

}

int main(int argc, const char **argv) {
    benchmark::Add add(argc, argv);
    add.runBenchmark();
    return 0;
}