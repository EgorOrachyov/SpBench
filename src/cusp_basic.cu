////////////////////////////////////////////////////////////////////////////////////
// MIT License                                                                    //
//                                                                                //
// Copyright (c) 2021 Egor Orachyov                                               //
//                                                                                //
// Permission is hereby granted, free of charge, to any person obtaining a copy   //
// of this software and associated documentation files (the "Software"), to deal  //
// in the Software without restriction, including without limitation the rights   //
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      //
// copies of the Software, and to permit persons to whom the Software is          //
// furnished to do so, subject to the following conditions:                       //
//                                                                                //
// The above copyright notice and this permission notice shall be included in all //
// copies or substantial portions of the Software.                                //
//                                                                                //
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     //
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       //
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    //
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         //
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  //
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  //
// SOFTWARE.                                                                      //
////////////////////////////////////////////////////////////////////////////////////

// The original example is https://cusplibrary.github.io/md_quickstart.html

#include <hip/hip_runtime.h>
#include <cusp_compiler_fence.hpp>

#include <cusp/hyb_matrix.h>
#include <cusp/monitor.h>
#include <cusp/gallery/poisson.h>
#include <cusp/krylov/cg.h>

// where to perform the computation
typedef cusp::device_memory MemorySpace;

// which floating point type to use
typedef float ValueType;

int main()
{
    // Version query
    int cuda_major =  CUDA_VERSION / 1000;
    int cuda_minor = (CUDA_VERSION % 1000) / 10;
    int thrust_major = THRUST_MAJOR_VERSION;
    int thrust_minor = THRUST_MINOR_VERSION;
    int cusp_major = CUSP_MAJOR_VERSION;
    int cusp_minor = CUSP_MINOR_VERSION;

    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
    std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;

    // create an empty sparse matrix structure (HYB format)
    cusp::hyb_matrix<int, ValueType, MemorySpace> A;

    // create a 2d Poisson problem on a 10x10 mesh
    cusp::gallery::poisson5pt(A, 10, 10);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<ValueType, MemorySpace> x(A.num_rows, 0);
    cusp::array1d<ValueType, MemorySpace> b(A.num_rows, 1);

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-3
    //  absolute_tolerance = 0
    //  verbose            = true
    cusp::monitor<ValueType> monitor(b, 100, 1e-3, 0, true);

    // set preconditioner (identity)
    cusp::identity_operator<ValueType, MemorySpace> M(A.num_rows, A.num_rows);

    // solve the linear system A * x = b with the Conjugate Gradient method
    cusp::krylov::cg(A, x, b, monitor, M);

    return 0;
}
