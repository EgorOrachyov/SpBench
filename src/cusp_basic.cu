//
// Created by Egor.Orachev on 26.01.2021.
//

// The original example is https://cusplibrary.github.io/md_quickstart.html

#include <hip/hip_runtime.h>
#include <cusp_compiler_fence.hpp>

#include <cusp/hyb_matrix.h>
#include <cusp/monitor.h>
#include <cusp/gallery/poisson.h>
#include <cusp/krylov/cg.h>

// where to perform the computation
typedef cusp::device_memory MemorySpace;

// which floating point type to use
typedef float ValueType;

int main()
{
    // Version query
    int cuda_major =  CUDA_VERSION / 1000;
    int cuda_minor = (CUDA_VERSION % 1000) / 10;
    int thrust_major = THRUST_MAJOR_VERSION;
    int thrust_minor = THRUST_MINOR_VERSION;
    int cusp_major = CUSP_MAJOR_VERSION;
    int cusp_minor = CUSP_MINOR_VERSION;

    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
    std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;

    // create an empty sparse matrix structure (HYB format)
    cusp::hyb_matrix<int, ValueType, MemorySpace> A;

    // create a 2d Poisson problem on a 10x10 mesh
    cusp::gallery::poisson5pt(A, 10, 10);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<ValueType, MemorySpace> x(A.num_rows, 0);
    cusp::array1d<ValueType, MemorySpace> b(A.num_rows, 1);

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-3
    //  absolute_tolerance = 0
    //  verbose            = true
    cusp::monitor<ValueType> monitor(b, 100, 1e-3, 0, true);

    // set preconditioner (identity)
    cusp::identity_operator<ValueType, MemorySpace> M(A.num_rows, A.num_rows);

    // solve the linear system A * x = b with the Conjugate Gradient method
    cusp::krylov::cg(A, x, b, monitor, M);

    return 0;
}
