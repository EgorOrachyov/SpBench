#include "hip/hip_runtime.h"
//
// Created by Egor.Orachev on 27.01.2021.
//

#include <benchmark_base.hpp>
#include <matrix_loader.hpp>
#include <args_processor.hpp>

#include <cusp_compiler_fence.hpp>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/multiply.h>
#include <cusp/elementwise.h>

#include <thrust/functional.h>

#define BENCH_DEBUG

namespace benchmark {

    template<typename T>
    struct logic_or
    {
        typedef T first_argument_type;
        typedef T second_argument_type;
        typedef T result_type;
        __thrust_exec_check_disable__
        __host__ __device__ T operator()(const T &lhs, const T &rhs) const {return lhs | rhs;}
    };

    template<typename T>
    struct logic_and
    {
        typedef T first_argument_type;
        typedef T second_argument_type;
        typedef T result_type;
        __thrust_exec_check_disable__
        __host__ __device__ T operator()(const T &lhs, const T &rhs) const {return lhs & rhs;}
    };

    typedef char value_type;
    static const value_type t = true;

    class Add: public BenchmarkBase {
    public:

        Add(int argc, const char** argv) {
            argsProcessor.parse(argc, argv);
            assert(argsProcessor.isParsed());

            benchmarkName = "Cusp-Add";
            experimentsCount = argsProcessor.getExperimentsCount();
        }

    protected:

        void setupBenchmark() override {

        }

        void tearDownBenchmark() override {

        }

        void setupExperiment(size_t experimentIdx, size_t &iterationsCount) override {
            auto& entry = argsProcessor.getEntries()[experimentIdx];

            iterationsCount = entry.iterations;

            const auto& file = entry.name;
            const auto& type = entry.isUndirected;

            MatrixLoader loader(file, type);
            loader.loadData();
            input = std::move(loader.getMatrix());

#ifdef BENCH_DEBUG
            std::cout << ">   Load A: \"" << file << "\" isUndirected: " << type << std::endl
                      << "                 size: " << input.nrows << " x " << input.ncols << " nvals: " << input.nvals << std::endl;
#endif // BENCH_DEBUG

            size_t n = input.nrows;
            assert(input.nrows == input.ncols);

            hostData = host_matrix_t(n, n, input.nvals);

            for (auto i = 0; i < input.nvals; i++) {
                hostData.row_indices[i] = input.rows[i];
                hostData.column_indices[i] = input.cols[i];
                hostData.values[i] = t;
            }

            A = std::move(device_matrix_t(hostData));

            thrust::identity<value_type> identity;
            logic_and<value_type> combine;
            logic_or<value_type> reduce;

            // compute M2 = M * M
            cusp::multiply(A, A, A2, identity, combine, reduce);
        }

        void tearDownExperiment(size_t experimentIdx) override {
            input = Matrix{};
            A = device_matrix_t{};
            A2 = device_matrix_t{};
        }

        void setupIteration(size_t experimentIdx, size_t iterationIdx) override {

        }

        void execIteration(size_t experimentIdx, size_t iterationIdx) override {
            // define multiply functors
            logic_or<value_type> reduce;

            // compute R = A + A2
            cusp::elementwise(A, A2, R, reduce);

#ifdef BENCH_DEBUG
            log << "   Result matrix: size " << R.num_rows << " x " << R.num_cols
                << " nvals " << R.num_entries << std::endl;
#endif
        }

        void tearDownIteration(size_t experimentIdx, size_t iterationIdx) override {
            R = device_matrix_t{};
        }

    protected:
        typedef cusp::coo_matrix<int, value_type, cusp::host_memory> host_matrix_t;
        typedef cusp::csr_matrix<int, value_type, cusp::device_memory> device_matrix_t;

        host_matrix_t hostData;
        device_matrix_t A;
        device_matrix_t A2;
        device_matrix_t R;

        ArgsProcessor argsProcessor;
        Matrix input;

    };

}

int main(int argc, const char** argv) {
    benchmark::Add add(argc, argv);
    add.runBenchmark();
    return 0;
}