#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////
// MIT License                                                                    //
//                                                                                //
// Copyright (c) 2021 Egor Orachyov                                               //
//                                                                                //
// Permission is hereby granted, free of charge, to any person obtaining a copy   //
// of this software and associated documentation files (the "Software"), to deal  //
// in the Software without restriction, including without limitation the rights   //
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      //
// copies of the Software, and to permit persons to whom the Software is          //
// furnished to do so, subject to the following conditions:                       //
//                                                                                //
// The above copyright notice and this permission notice shall be included in all //
// copies or substantial portions of the Software.                                //
//                                                                                //
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     //
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       //
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    //
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         //
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  //
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  //
// SOFTWARE.                                                                      //
////////////////////////////////////////////////////////////////////////////////////

#include <benchmark_base.hpp>
#include <matrix_loader.hpp>
#include <args_processor.hpp>

#include <cusp_compiler_fence.hpp>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/multiply.h>
#include <cusp/elementwise.h>

#include <thrust/functional.h>

#define BENCH_DEBUG

namespace benchmark {

    template<typename T>
    struct logic_or
    {
        typedef T first_argument_type;
        typedef T second_argument_type;
        typedef T result_type;
        __thrust_exec_check_disable__
        __host__ __device__ T operator()(const T &lhs, const T &rhs) const {return lhs | rhs;}
    };

    template<typename T>
    struct logic_and
    {
        typedef T first_argument_type;
        typedef T second_argument_type;
        typedef T result_type;
        __thrust_exec_check_disable__
        __host__ __device__ T operator()(const T &lhs, const T &rhs) const {return lhs & rhs;}
    };

    typedef char value_type;
    static const value_type t = true;

    class MultiplyAdd: public BenchmarkBase {
    public:

        MultiplyAdd(int argc, const char** argv) {
            argsProcessor.parse(argc, argv);
            assert(argsProcessor.isParsed());

            benchmarkName = "Cusp-Multiply-Add";
            experimentsCount = argsProcessor.getExperimentsCount();
        }

    protected:

        void setupBenchmark() override {

        }

        void tearDownBenchmark() override {

        }

        void setupExperiment(size_t experimentIdx, size_t &iterationsCount, std::string& name) override {
            auto& entry = argsProcessor.getEntries()[experimentIdx];

            iterationsCount = entry.iterations;
            name = entry.name;

            const auto& file = entry.name;
            const auto& type = entry.isUndirected;

            MatrixLoader loader(file, type);
            loader.loadData();
            input = std::move(loader.getMatrix());

#ifdef BENCH_DEBUG
            log       << ">   Load matrix: \"" << file << "\" isUndirected: " << type << std::endl
                      << "                 size: " << input.nrows << " x " << input.ncols << " nvals: " << input.nvals << std::endl;
#endif // BENCH_DEBUG

            size_t n = input.nrows;
            assert(input.nrows == input.ncols);

            hostData = host_matrix_t(n, n, input.nvals);

            for (auto i = 0; i < input.nvals; i++) {
                hostData.row_indices[i] = input.rows[i];
                hostData.column_indices[i] = input.cols[i];
                hostData.values[i] = t;
            }

            matrix = std::move(device_matrix_t(hostData));
        }

        void tearDownExperiment(size_t experimentIdx) override {
            input = Matrix{};
            matrix = device_matrix_t{};
        }

        void setupIteration(size_t experimentIdx, size_t iterationIdx) override {

        }

        void execIteration(size_t experimentIdx, size_t iterationIdx) override {
            // define multiply functors
            thrust::identity<value_type> identity;
            logic_and<value_type> combine;
            logic_or<value_type> reduce;

            // compute R = M * M
            cusp::multiply(matrix, matrix, R, identity, combine, reduce);

            // compute R = R + M
            cusp::elementwise(matrix, R, R, reduce);

#ifdef BENCH_DEBUG
            log << "   Result matrix: size " << R.num_rows << " x " << R.num_cols
                << " nvals " << R.num_entries << std::endl;
#endif
        }

        void tearDownIteration(size_t experimentIdx, size_t iterationIdx) override {
            R = device_matrix_t{};
        }

    protected:
        typedef cusp::coo_matrix<int, value_type, cusp::host_memory> host_matrix_t;
        typedef cusp::csr_matrix<int, value_type, cusp::device_memory> device_matrix_t;

        host_matrix_t hostData;
        device_matrix_t matrix;
        device_matrix_t R;

        ArgsProcessor argsProcessor;
        Matrix input;

    };

}

int main(int argc, const char** argv) {
    benchmark::MultiplyAdd multiplyAdd(argc, argv);
    multiplyAdd.runBenchmark();
    return 0;
}